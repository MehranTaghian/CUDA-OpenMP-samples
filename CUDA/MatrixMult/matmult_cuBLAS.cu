#include <hip/hip_runtime.h>

#include<hipblas.h>
#include<cstdlib>
#include<hiprand/hiprand.h>
#include<iostream>
#include<ctime>


void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda = m, ldb = k, ldc = m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start;
    hipEventCreate(&start);

    hipEvent_t stop;
    hipEventCreate(&stop);

    hipEventRecord(start, NULL);

    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);


    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    printf("Elapsed time in msec = %f\n", msecTotal);

    // Destroy the handle
    hipblasDestroy(handle);

}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {
    for (int i = 0; i < nr_rows_A; ++i) {
        for (int j = 0; j < nr_cols_A; ++j) {
            std::cout << A[j * nr_rows_A + i] << " ";

        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
    // Allocate 3 arrays on CPU
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

    int n = 0;
    printf("Enter dimension:");
    scanf("%u", &n);


    // for simplicity we are going to use square arrays
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = n;

    float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    // Allocate 3 arrays on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, nr_rows_A * nr_cols_A * sizeof(float));
    hipMalloc(&d_B, nr_rows_B * nr_cols_B * sizeof(float));
    hipMalloc(&d_C, nr_rows_C * nr_cols_C * sizeof(float));

    // Fill the arrays A and B on GPU with random numbers
    GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
    GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

    // Optionally we can copy the data back on CPU and print the arrays
    hipMemcpy(h_A, d_A, nr_rows_A * nr_cols_A * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, nr_rows_B * nr_cols_B * sizeof(float), hipMemcpyDeviceToHost);


    // Multiply A and B on GPU
    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

    // Copy (and print) the result on host memory
    hipMemcpy(h_C, d_C, nr_rows_C * nr_cols_C * sizeof(float), hipMemcpyDeviceToHost);

    /*std::cout << "A =" << std::endl;
    print_matrix(h_A, nr_rows_A, nr_cols_A);
    std::cout << "B =" << std::endl;
    print_matrix(h_B, nr_rows_B, nr_cols_B);
    std::cout << "C =" << std::endl;
    print_matrix(h_C, nr_rows_C, nr_cols_C);*/

    //Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free CPU memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}