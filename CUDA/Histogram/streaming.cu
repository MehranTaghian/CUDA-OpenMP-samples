#include "hip/hip_runtime.h"

//#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include <omp.h>
#include<iostream>
#define MAX_HISTORGRAM_NUMBER 10000
#define ARRAY_SIZE 102400000

#define CHUNK_SIZE 100
#define THREAD_COUNT 512
#define SCALER 80
hipError_t histogramWithCuda(int *a, unsigned long long int *c, int block_size, int thread_count, int chunk_size, int scalar);

__global__ void histogramKernelSingle(unsigned long long int *c, int *a, int chunk_size, int scalar)
{
    unsigned long long int worker = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned long long int start = worker * chunk_size;
    unsigned long long int end = start + chunk_size;
    for (int ex = 0; ex < scalar; ex++)
        for (long long int i = start; i < end; i++)
        {
            if (i < ARRAY_SIZE)
                atomicAdd(&c[a[i]], 1);
            else
            {
                break;
            }
        }

}
int main()
{
    int thread_count = 32;
    int chunk_size = 100000;
    int grid_size = 32;
    int scalar = 20;

    printf("Enter thread count: ");
    scanf("%u", &thread_count);
    printf("Enter chunk size: ");
    scanf("%u", &chunk_size);
    //chunk_size = CHUNK_SIZE;
    printf("Enter grid size: ");
    scanf("%u", &grid_size);



    int* a;
    hipHostMalloc((void**)&a, sizeof(int)*ARRAY_SIZE);

    unsigned long long int* c;
    hipHostMalloc((void**)&c, sizeof(unsigned long long int) * MAX_HISTORGRAM_NUMBER);

    for (unsigned long long i = 0; i < ARRAY_SIZE; i++)
        a[i] = rand() % MAX_HISTORGRAM_NUMBER;
    for (unsigned long long i = 0; i < MAX_HISTORGRAM_NUMBER; i++)
        c[i] = 0;


    // Add vectors in parallel.
    double start_time = omp_get_wtime();
    hipError_t cudaStatus = histogramWithCuda(a, c, grid_size, thread_count, chunk_size, scalar);
    double end_time = omp_get_wtime();
    std::cout << "Elapsed Time: " << end_time - start_time;
    // =
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    unsigned long long int R = 0;
    for (int i = 0; i < MAX_HISTORGRAM_NUMBER; i++)
    {
        R += c[i];
        //		printf("%d	", c[i]);
    }
    printf("\nCORRECT:%ld	", R / (SCALER));
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t histogramWithCuda(int *a, unsigned long long int *c, int grid_size, int thread_count, int chunk_size, int scalar)
{
    const int num_stream = 4;
    hipStream_t streams[num_stream];

    int *dev_a = 0;
    unsigned long long int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, ARRAY_SIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    for (int i = 0; i < num_stream; i++) {
        //hipStream_t stream;

        hipStreamCreate(&streams[i]);

        histogramKernelSingle << <grid_size, thread_count, 0, streams[i] >> > (dev_c, dev_a, chunk_size, scalar);
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpyAsync(c, dev_c, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyDeviceToHost, streams[i]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    return cudaStatus;

}
