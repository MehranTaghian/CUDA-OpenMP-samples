#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, int blockSize, int gridSize);
void fillMat(int * v, int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 1024 * 1024;
    const int block_size = 1024;
    const int grid_size = arraySize / block_size;

    int* a = (int*)malloc(arraySize * sizeof(int));
    int* b = (int*)malloc(arraySize * sizeof(int));
    int* c = (int*)malloc(arraySize * sizeof(int));

    fillMat(a, arraySize);
    fillMat(b, arraySize);


    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize, block_size, grid_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


void fillMat(int * v, int size) {
    static int L = 0;
    for (int i = 0; i < size; i++) {
        v[i] = L++;
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, int blockSize, int gridSize)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <gridSize, blockSize >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipEventRecord(stop, NULL);
    hipError_t error = hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    printf("elapsed time is: %f\n", msecTotal);

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
