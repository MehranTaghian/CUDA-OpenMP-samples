#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdlib.h>
#include <stdio.h>


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int n, unsigned int matSizeX, unsigned int matSizeY);
void fillMat(int * v, int matSizeX, int matSizeY);
void printMat(int * v, int matSizeX, int matSizeY);
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    c[i*blockDim.y + j] = a[i*blockDim.y + j] + b[i*blockDim.y + j];
}

int main()
{
    const int n = 10000;
    const int matSizeX = 32;
    const int matSizeY = 32;

    int * a;
    int * b;
    int * c;
    a = (int*)malloc(sizeof(int)*matSizeX*matSizeY*n);
    b = (int*)malloc(sizeof(int)*matSizeX*matSizeY*n);
    c = (int*)malloc(sizeof(int)*matSizeX*matSizeY*n);

    fillMat(a, matSizeX, matSizeY);
    fillMat(b, matSizeX, matSizeY);


    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, n, matSizeX, matSizeY);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    //printMat(a, matSizeX, matSizeY);
    //printMat(b, matSizeX, matSizeY);
    //printMat(c, matSizeX, matSizeY);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
void fillMat(int * v, int matSizeX, int matSizeY) {
    static int L = 0;
    for (int i = 0; i < matSizeX; i++) {
        for (int j = 0; j < matSizeY; j++)
            v[i*matSizeY + j] = L++;


    }
}
void printMat(int * v, int matSizeX, int matSizeY) {
    int i;
    printf("[-] Vector elements: \n");
    for (int i = 0; i < matSizeX; i++) {
        for (int j = 0; j < matSizeY; j++)
            printf("%d	", v[i*matSizeY + j]);
        printf("\n");

    }
    printf("\b\b  \n");
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int n, unsigned int matSizeX, unsigned int matSizeY)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

    hipEventRecord(start, NULL);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, matSizeX*matSizeY * n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_a, matSizeX*matSizeY * n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_b, matSizeX*matSizeY * n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, matSizeX*matSizeY * n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, matSizeX*matSizeY * n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    dim3 block_size = dim3(matSizeX, matSizeY, 1);
    dim3 grid_size = dim3(n, 1, 1);
    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <grid_size, block_size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, matSizeX*matSizeY * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipEventRecord(stop, NULL);
    hipError_t error = hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    printf("elapsed time is: %f\n", msecTotal);

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
